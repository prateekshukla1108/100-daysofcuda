#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#define GRID_SIZE 32
#define NUM_CELLS (GRID_SIZE * GRID_SIZE)
#define NUM_RESOURCE_TYPES 2
#define NUM_AGENTS 1024
#define NUM_ITERATIONS 1000
#define STARVATION_LIMIT 50

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if(err != hipSuccess) { \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

struct __align__(16) ResourceCell {
    float resource[NUM_RESOURCE_TYPES];
    float maxResource[NUM_RESOURCE_TYPES];
    float baseRegen[NUM_RESOURCE_TYPES];
};

struct __align__(16) Agent {
    int x;
    int y;
    int type;
    float desired[NUM_RESOURCE_TYPES];
    float totalReward;
    int alive;
    int starvationCounter;
};

__device__ float atomicExtract(float *address, float desiredVal) {
    int* address_as_int = (int*)address;
    int old_int = *address_as_int;
    float old = __int_as_float(old_int);
    float extracted;
    float new_val;
    do {
        if(old < desiredVal) {
            extracted = old;
            new_val = 0.0f;
        } else {
            extracted = desiredVal;
            new_val = old - desiredVal;
        }
        int new_int = __float_as_int(new_val);
        int prev_int = atomicCAS(address_as_int, old_int, new_int);
        if(prev_int == old_int) break;
        old_int = prev_int;
        old = __int_as_float(old_int);
    } while(true);
    return extracted;
}

__global__ void initAgentKernel(Agent *agents, hiprandState *agentStates, unsigned int seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= NUM_AGENTS) return;
    hiprand_init(seed, id, 0, &agentStates[id]);
    hiprandState localState = agentStates[id];
    agents[id].x = hiprand(&localState) % GRID_SIZE;
    agents[id].y = hiprand(&localState) % GRID_SIZE;
    agents[id].type = hiprand(&localState) % 2;
    for(int i = 0; i < NUM_RESOURCE_TYPES; i++){
        agents[id].desired[i] = 1.0f + hiprand_uniform(&localState);
    }
    agents[id].totalReward = 0.0f;
    agents[id].alive = 1;
    agents[id].starvationCounter = 0;
    agentStates[id] = localState;
}

__global__ void initResourceKernel(ResourceCell *cells, hiprandState *cellStates, unsigned int seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= NUM_CELLS) return;
    hiprand_init(seed, id, 0, &cellStates[id]);
    hiprandState localState = cellStates[id];
    for(int i = 0; i < NUM_RESOURCE_TYPES; i++){
        cells[id].maxResource[i] = 10.0f + 10.0f * hiprand_uniform(&localState);
        cells[id].resource[i] = cells[id].maxResource[i] * hiprand_uniform(&localState);
        cells[id].baseRegen[i] = 0.1f + 0.2f * hiprand_uniform(&localState);
    }
    cellStates[id] = localState;
}

__global__ void agentKernel(Agent *agents, ResourceCell *cells, hiprandState *agentStates) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= NUM_AGENTS) return;
    hiprandState state = agentStates[id];
    if(agents[id].alive) {
        int cellIdx = agents[id].x + agents[id].y * GRID_SIZE;
        float harvested[NUM_RESOURCE_TYPES];
        int insufficient = 1;
        for(int i = 0; i < NUM_RESOURCE_TYPES; i++){
            harvested[i] = atomicExtract(&cells[cellIdx].resource[i], agents[id].desired[i]);
            agents[id].totalReward += harvested[i];
            if(harvested[i] >= 0.5f * agents[id].desired[i]) {
                insufficient = 0;
            }
        }
        if(insufficient)
            agents[id].starvationCounter++;
        else if(agents[id].starvationCounter > 0)
            agents[id].starvationCounter--;
        if(agents[id].starvationCounter > STARVATION_LIMIT)
            agents[id].alive = 0;
        float rdx = hiprand_uniform(&state);
        float rdy = hiprand_uniform(&state);
        int dx = (rdx < (1.0f/3.0f)) ? -1 : (rdx < (2.0f/3.0f) ? 0 : 1);
        int dy = (rdy < (1.0f/3.0f)) ? -1 : (rdy < (2.0f/3.0f) ? 0 : 1);
        int newX = (agents[id].x + dx + GRID_SIZE) % GRID_SIZE;
        int newY = (agents[id].y + dy + GRID_SIZE) % GRID_SIZE;
        agents[id].x = newX;
        agents[id].y = newY;
        int newCell = newX + newY * GRID_SIZE;
        for(int i = 0; i < NUM_RESOURCE_TYPES; i++){
            float clampVal = 0.5f * cells[newCell].maxResource[i];
            if(agents[id].desired[i] > clampVal)
                agents[id].desired[i] = clampVal;
        }
    }
    agentStates[id] = state;
}

__global__ void regenKernel(ResourceCell *cells, hiprandState *cellStates) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= NUM_CELLS) return;
    hiprandState state = cellStates[id];
    for(int i = 0; i < NUM_RESOURCE_TYPES; i++){
        float current = cells[id].resource[i];
        float maxVal = cells[id].maxResource[i];
        float regenFactor = cells[id].baseRegen[i] * (1.0f - current / maxVal);
        float noise = 0.05f * (hiprand_uniform(&state) - 0.5f);
        float regen = regenFactor + noise;
        float depletion = 0.01f * current;
        float newVal = current + regen - depletion;
        if(newVal < 0.0f) newVal = 0.0f;
        if(newVal > maxVal) newVal = maxVal;
        cells[id].resource[i] = newVal;
    }
    cellStates[id] = state;
}

int main(){
    Agent *d_agents;
    ResourceCell *d_cells;
    hiprandState *d_agentStates, *d_cellStates;
    CHECK_CUDA(hipMalloc(&d_agents, NUM_AGENTS * sizeof(Agent)));
    CHECK_CUDA(hipMalloc(&d_cells, NUM_CELLS * sizeof(ResourceCell)));
    CHECK_CUDA(hipMalloc(&d_agentStates, NUM_AGENTS * sizeof(hiprandState)));
    CHECK_CUDA(hipMalloc(&d_cellStates, NUM_CELLS * sizeof(hiprandState)));
    
    int blockSize = 256;
    int numAgentBlocks = (NUM_AGENTS + blockSize - 1) / blockSize;
    int numCellBlocks = (NUM_CELLS + blockSize - 1) / blockSize;
    unsigned int seed = (unsigned int) time(NULL);
    
    initAgentKernel<<<numAgentBlocks, blockSize>>>(d_agents, d_agentStates, seed);
    CHECK_CUDA(hipGetLastError());
    initResourceKernel<<<numCellBlocks, blockSize>>>(d_cells, d_cellStates, seed);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    
    for(int iter = 0; iter < NUM_ITERATIONS; iter++){
        agentKernel<<<numAgentBlocks, blockSize>>>(d_agents, d_cells, d_agentStates);
        CHECK_CUDA(hipGetLastError());
        regenKernel<<<numCellBlocks, blockSize>>>(d_cells, d_cellStates);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipDeviceSynchronize());
        
        if(iter % 100 == 0 || iter == NUM_ITERATIONS - 1) {
            Agent *h_agentsTemp = (Agent*)malloc(NUM_AGENTS * sizeof(Agent));
            ResourceCell *h_cellsTemp = (ResourceCell*)malloc(NUM_CELLS * sizeof(ResourceCell));
            CHECK_CUDA(hipMemcpy(h_agentsTemp, d_agents, NUM_AGENTS * sizeof(Agent), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(h_cellsTemp, d_cells, NUM_CELLS * sizeof(ResourceCell), hipMemcpyDeviceToHost));
            
            int aliveCount = 0;
            float totalReward = 0.0f;
            for(int i = 0; i < NUM_AGENTS; i++){
                if(h_agentsTemp[i].alive){
                    aliveCount++;
                    totalReward += h_agentsTemp[i].totalReward;
                }
            }
            float avgReward = (aliveCount > 0) ? totalReward / aliveCount : 0.0f;
            float avgResource[NUM_RESOURCE_TYPES] = {0.0f};
            for (int i = 0; i < NUM_CELLS; i++){
                for (int j = 0; j < NUM_RESOURCE_TYPES; j++){
                    avgResource[j] += h_cellsTemp[i].resource[j];
                }
            }
            for(int j = 0; j < NUM_RESOURCE_TYPES; j++){
                avgResource[j] /= NUM_CELLS;
            }
            printf("Iteration %d: Alive Agents = %d, Avg Reward = %f, Avg Resources = ", iter, aliveCount, avgReward);
            for(int j = 0; j < NUM_RESOURCE_TYPES; j++){
                printf("%f ", avgResource[j]);
            }
            printf("\n");
            free(h_agentsTemp);
            free(h_cellsTemp);
        }
    }
    
    Agent *h_agents = (Agent*)malloc(NUM_AGENTS * sizeof(Agent));
    ResourceCell *h_cells = (ResourceCell*)malloc(NUM_CELLS * sizeof(ResourceCell));
    CHECK_CUDA(hipMemcpy(h_agents, d_agents, NUM_AGENTS * sizeof(Agent), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_cells, d_cells, NUM_CELLS * sizeof(ResourceCell), hipMemcpyDeviceToHost));
    
    float totalReward = 0.0f;
    int aliveCount = 0;
    for(int i = 0; i < NUM_AGENTS; i++){
        totalReward += h_agents[i].totalReward;
        if(h_agents[i].alive)
            aliveCount++;
    }
    printf("Final: Total Reward = %f, Alive Agents = %d\n", totalReward, aliveCount);
    
    free(h_agents);
    free(h_cells);
    CHECK_CUDA(hipFree(d_agents));
    CHECK_CUDA(hipFree(d_cells));
    CHECK_CUDA(hipFree(d_agentStates));
    CHECK_CUDA(hipFree(d_cellStates));
    return 0;
}

